#include "hip/hip_runtime.h"
#include <cmath>
#include <algorithm> 
#include <iostream>
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
 // imports do thrust
#include <thrust/host_vector.h>0000000
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h> 
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;
int main(){
    int N, M;
    cin >> N >> M;
    cout << N << endl;
    cout << M << endl;
    thrust::host_vector<int>filmes_por_categoria_cpu(M);
    thrust::host_vector<int> start_times_cpu(N);
    thrust::host_vector<int> end_times_cpu(N);
    thrust::host_vector<int> categories_cpu(N);

    for (int i = 0; i < M; i++){
        cin >> filmes_por_categoria_cpu[i];
    }

    for (int i = 0; i < N; i++){
        cin >> start_times_cpu[i];
        cin >> end_times_cpu[i];
        cin >> categories_cpu[i];
        if (start_times_cpu[i] == 0){
            start_times_cpu[i] = 24;
        }
        if (end_times_cpu[i] == 0){
            end_times_cpu[i] = 24;
        }
        if (start_times_cpu[i] < 0){
            continue;
        }
        if (end_times_cpu[i] < 0){
            continue;
        }
    }

    thrust::device_vector<int> start_times_gpu(start_times_cpu);
    thrust::device_vector<int> end_times_gpu(end_times_cpu);
    thrust::device_vector<int> categories_gpu(categories_cpu);
    thrust::device_vector<int> filmes_por_categoria_gpu(filmes_por_categoria_cpu);


    thrust::device_vector<int> dp(N * M, 0);

    // Inicializar a primeira linha da matriz com zeros
    thrust::fill(dp.begin(), dp.begin() + M, 0);

    // Preencher a matriz com as soluções para subproblemas menores
    // Preencher a matriz com as soluções para subproblemas menores
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
            int max_count = 0;
            for (int k = 0; k < i; k++) {
              if (categories_gpu[k] == j && end_times_gpu[k] <= start_times_gpu[i] && dp[(k*(M)) + j] + 1 <= filmes_por_categoria_gpu[j]) {
                  max_count = max(max_count, dp[(k*(M)) + j] + 1);
              } else {
                  max_count = max(max_count, dp[(k*(M)) + j]);
              }
            }
            dp[(i*(M)) + j] = max_count;
        }
    }

    // Encontrar o número máximo de filmes que podem ser assistidos
    int max_count = 0;
    int max_j = 0;
    for (int j = 0; j < M; j++) {
        if (dp[(N-1 * (M)) + j] > max_count) {
            max_count = dp[(N-1 * (M)) + j];
            max_j = j;
        }
    }



    cout << max_count << endl;
    cout << max_j << endl;
    cout << dp[(N-1 * (M)) + max_j] << endl;

    for (int i = 0; i < N*M; i++) {
        cout << dp[i] << " ";
    }



    return 0;
}



// O primeiro loop `for` no código está preenchendo a matriz `dp` com as soluções para subproblemas menores. Ele itera sobre os valores de `i` de 0 a N-1 e `j` de 0 a M-1, representando as posições da matriz `dp`.

// Dentro desse loop, ele calcula o número máximo de filmes que podem ser assistidos até o filme i e categoria j. Isso é feito comparando as informações dos filmes anteriores (de 0 a i-1) e verificando se as condições são atendidas para adicionar um novo filme.

// A condição para adicionar um novo filme é a seguinte:
// - A categoria do filme k (com k de 0 a i-1) é igual a j.
// - O horário de término do filme k é menor ou igual ao horário de início do filme i.
// - O número total de filmes na categoria j até o momento, representado por `dp[(k*(M)) + j]`, mais 1, é menor ou igual ao número máximo de filmes permitidos na categoria j, representado por `filmes_por_categoria_gpu[j]`.

// Se todas essas condições forem atendidas, o número máximo de filmes é atualizado para `dp[(k*(M)) + j] + 1`. Caso contrário, o número máximo de filmes permanece o mesmo, `dp[(k*(M)) + j]`.

// No final do primeiro loop `for`, a matriz `dp` estará preenchida com as soluções ótimas para cada subproblema, ou seja, o número máximo de filmes que podem ser assistidos até cada posição da matriz.

// Após o primeiro loop `for`, o código encontra o número máximo de filmes que podem ser assistidos verificando o valor mais alto na última linha da matriz `dp`, correspondente aos filmes N-1 e todas as categorias.

// Por fim, o código imprime o número máximo de filmes e também imprime a matriz `dp` para fins de depuração.

// Desculpe pela confusão anterior. Houve um erro de interpretação no código. Vamos esclarecer o significado dos elementos da matriz e como encontrar o número máximo de filmes que podem ser assistidos em um dia.

// No código fornecido, a matriz `dp` não representa o número máximo de filmes que podem ser assistidos para cada categoria. Na verdade, a matriz `dp` é usada para calcular a quantidade máxima de filmes que podem ser assistidos considerando todas as combinações possíveis de filmes e categorias até um determinado ponto.

// Cada elemento `dp[i*(M) + j]` da matriz `dp` representa o número máximo de filmes que podem ser assistidos considerando os primeiros `i` filmes e as primeiras `j` categorias. Portanto, o valor em `dp[N*(M) + j]` representa o número máximo de filmes que podem ser assistidos considerando todos os filmes e a categoria `j`.

// Para encontrar o número máximo de filmes que podem ser assistidos em um dia, considerando todas as categorias, é necessário encontrar o maior valor entre `dp[N*(M) + j]` para `j` variando de 0 a M-1.

// Em relação aos horários, o código já realiza a verificação das condições `end_times[k] <= start_times[i]` para garantir que os horários não se sobreponham. Portanto, os filmes só serão considerados se o horário de término do filme anterior for menor ou igual ao horário de início do filme atual.

// No entanto, o código atual não considera a restrição de não assistir mais de um filme ao mesmo tempo. Se você deseja garantir que os horários não estejam se sobrepondo, será necessário modificar o código para levar em conta essa restrição adicional.

// Uma abordagem possível seria usar programação dinâmica com uma matriz tridimensional, onde o terceiro índice representaria o horário de início. Cada elemento `dp[i][j][k]` representaria o número máximo de filmes que podem ser assistidos considerando os primeiros `i` filmes, as primeiras `j` categorias e o horário de início `k`. Você teria que ajustar o código para essa nova estrutura e considerar as restrições de horário ao calcular as soluções.

// Espero que isso esclareça a sua dúvida. Se precisar de mais informações, fique à vontade para perguntar!