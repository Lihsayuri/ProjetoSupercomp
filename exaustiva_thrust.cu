#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

struct Filme{
    int inicio;
    int fim;
    int categoria;
};

struct StructSchedule{
    vector<int> filmes;
    int qtd_filmes;
};

struct FilmeProcessado{
    int categoria;
    bitset<24> horario;
};

void preenche_bitset(int &horarios_disponiveis, int inicio, int fim){
    cout << inicio << " " << fim << endl; 
    //cout << horarios_disponiveis.size() << endl;
    for (int i = 0; i < 24; i++){
        if (i >= inicio && i < fim){
            horarios_disponiveis  |= (1 << i);
        }
        else if (inicio > fim && (i >= inicio || i < fim)){
            horarios_disponiveis |= (1 << i);
        }
        else if(inicio == fim){
            horarios_disponiveis |= (1 << i);
        }
    }
}

int and_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    for (size_t i = 0; i < v1.size(); i++) {
        if (v1[i] && v2[i]) {
            return 1;
        }
    }
    return 0;
}

vector<bool> or_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    vector<bool> result(v1.size());
    for (size_t i = 0; i < v1.size(); i++) {
        result[i] = v1[i] || v2[i];
    }
    return result;
}

struct busca_exaustiva_gpu 
{
    int config;   
    int qtd_filmes;
    vector<int> &filmes_por_categoria; 
    busca_exaustiva_gpu(int config_, int qtd_filmes_, vector<int> &filmes_por_categoria_) : config(config_), qtd_filmes(qtd_filmes_), filmes_por_categoria(filmes_por_categoria_) {}
    __host__ __device__
    int operator()(const vector<int> &categoria_filmes, const vector<vector<bool>> &horario_filmes) {
        vector<bool> horarios_disponiveis(24, false);
        vector<int> filmes_por_categoria_aux = filmes_por_categoria;
        int max_count = 0;
        for (int i = 0; i < qtd_filmes; i++){
            if (config & (1 << i)){
                if (filmes_por_categoria_aux[categoria_filmes[i]-1] > 0){
                    int horario_analisado = and_vectors(horarios_disponiveis, horario_filmes[i]);
                    // vector<bool> horario_analisado = horarios_disponiveis & vetor_filmes_processado[i].horario;
                    if ((horario_analisado != 0)) return -1;
                    filmes_por_categoria_aux[categoria_filmes[i]-1]--;
                    horarios_disponiveis = or_vectors(horarios_disponiveis, horario_filmes[i]);
                    max_count += 1;
                }
                else{
                    return -1;
                }
            }
        
        }

        return max_count;
    }
};



int main(){
    int qtd_filmes, qtd_categorias;
    cin >> qtd_filmes >> qtd_categorias;

    vector<int> filmes_por_categoria(qtd_categorias, 0);
    Filme filme_vazio = {0, 0, 0};
    vector<Filme> vetor_filmes (qtd_filmes, filme_vazio);

    for (int i = 0; i < qtd_categorias; i++){
        cin >> filmes_por_categoria[i];
    }

    for (int i = 0; i < qtd_filmes; i++){
        Filme filme;
        cin >> filme.inicio >> filme.fim >> filme.categoria;
        if (filme.inicio == 0) filme.inicio = 24;
        if (filme.fim == 0) filme.fim = 24;
        if (filme.inicio < 0 || filme.fim < 0) continue;

        vetor_filmes[i] = filme;
    }

    thrust::host_vector<int> categoria_filmes(qtd_filmes);
    thrust::host_vector<int> horarios_filmes_cpu(qtd_filmes); 


    for (int i = 0; i < qtd_filmes; i++){
        horarios_filmes_cpu[i] = 0;
        preenche_bitset(horarios_filmes_cpu[i], vetor_filmes[i].inicio-1, vetor_filmes[i].fim-1);
        categoria_filmes[i] = vetor_filmes[i].categoria;
    }



    thrust::device_vector<int> config_vector_gpu(pow(2, qtd_filmes));

    thrust::sequence(config_vector_gpu.begin(), config_vector_gpu.end());

    thrust::device_vector<int> categoria_filmes_gpu(categoria_filmes);
    thrust::device_vector<int> horarios_filmes_gpu(horarios_filmes_cpu);


    for (int i = 0; i < qtd_filmes; i++){
        cout << horarios_filmes_cpu[i] << endl;
    }

    // for (int i = 0; i < pow(2, qtd_filmes); i++){
    //     thrust::transform(vetor_filmes_processado.begin(), vetor_filmes_processado.end(), filmes_por_categoria.begin(), configs_aceitas_gpu.begin(), busca_exaustiva_gpu(config_vector_gpu[i], qtd_filmes));
    // }

    // // thrust::transform(config_vector_gpu, config_vector_gpu, config_vector_gpu.begin(), busca_exaustiva_gpu);

    // thrust::host_vector<int> config_vector_cpu_final = config_vector_gpu;

    // int max_count = 0;
    // for (int i = 0; i < pow(2, qtd_filmes); i++){
    //     if (config_vector_cpu[i] > max_count){
    //         max_count = config_vector_cpu[i];
    //     }
    // }

    // cout << max_count << endl;
}




// g++ -Wl,-z,stack-size=4194304 exaustiva.cpp -o exaustiva
//  g++ -Wl,-z,stack-size=6000000000 -fopenmp exaustiva.cpp -o exaustiva
// user@monstrinho:~/ProjetoSupercomp$ ./exaustiva 
// nvcc -arch=sm_70 -rdc=true -o exaustiva_thrust exaustiva_thrust.cu