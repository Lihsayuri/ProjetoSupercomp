#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
 // imports do thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h> 
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

int main(){
    int N, M;
    cin >> N >> M;

    thrust::device_vector<int> start_times(N);
    thrust::device_vector<int> end_times(N);
    thrust::device_vector<int> categories(N);
    thrust::device_vector<int> filmes_por_categoria(M, 0);

    for (int i = 0; i < M; i++){
        cin >> filmes_por_categoria[i];
    }

    for (int i = 0; i < N; i++){
        cin >> start_times[i];
        cin >> end_times[i];
        cin >> categories[i];
        if (start_times[i] == 0){
            start_times[i] = 24;
        }
        if (end_times[i] == 0){
            end_times[i] = 24;
        }
        if (start_times[i] < 0){
            continue;
        }
        if (end_times[i] < 0){
            continue;
        }
    }

    thrust::device_vector<int> dp((N+1) * (M+1), 0);

    // Inicializar a primeira linha da matriz com zeros
    thrust::fill(dp.begin(), dp.begin() + M + 1, 0);

    // Preencher a matriz com as soluções para subproblemas menores
    // Preencher a matriz com as soluções para subproblemas menores
    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= M; j++) {
            // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
            int max_count = 0;
            for (int k = 0; k < i; k++) {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= L[j-1]) {
                max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
            } else {
                max_count = max(max_count, dp[(k*(M+1)) + j]);
            }
            }
            dp[(i*(M+1)) + j] = max_count;
        }
    }

    // Encontrar o número máximo de filmes que podem ser assistidos
    int max_count = 0;
    int max_j = 0;
    for (int j = 1; j <= M; j++) {
        if (dp[(N * (M + 1)) + j] > max_count) {
            max_count = dp[(N * (M + 1)) + j];
            max_j = j;
        }
    }

    while (i > 0 && j > 0) {
        if (categories[i - 1] == j && end_times[i - 1] <= start_times[i] && dp[(i - 1) * (M + 1) + j - 1] + 1 == dp[i * (M + 1) + j]) {
            filmes_selecionados.push_back(i);
            i--;
            j--;
        } else {
            i--;
        }
    }

    // Os filmes selecionados estão armazenados em ordem inversa, então é necessário inverter a ordem
    std::reverse(filmes_selecionados.begin(), filmes_selecionados.end());

    // Imprimir os filmes selecionados
    std::cout << "Filmes selecionados: ";
    for (int filme : filmes_selecionados) {
        std::cout << filme << " ";
    }
    std::cout << std::endl;

    return 0;

}


// g++ -Wl,-z,stack-size=4194304 exaustiva.cpp -o exaustiva
//  g++ -Wl,-z,stack-size=6000000000 -fopenmp exaustiva.cpp -o exaustiva
// user@monstrinho:~/ProjetoSupercomp$ ./exaustiva 