#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

struct Filme{
    int inicio;
    int fim;
    int categoria;
};

struct FilmeProcessado{
    int categoria;
    vector<bool> horario;
};

struct StructSchedule{
    vector<int> filmes;
    int qtd_filmes;
};


void preenche_bitset(vector<bool> &horarios_disponiveis, int inicio, int fim){
    for (int i = 0; i < 24; i++){
        if (i >= inicio && i < fim){
            horarios_disponiveis[i] = true;
        }
        else if (inicio > fim && (i >= inicio || i < fim)){
            horarios_disponiveis[i] = true;
        }
        else if(inicio == fim){
            horarios_disponiveis[i] = true;
        }
    }
}

struct saxpy
{
    int a;    
    saxpy(int a_) : a(a_) {};
    __host__ __device__
    double operator()(const int& x, const int& y) {
           return a * x + y;
    }
};

int and_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    for (size_t i = 0; i < v1.size(); i++) {
        if (v1[i] && v2[i]) {
            return 1;
        }
    }
    return 0;
}

vector<bool> or_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    vector<bool> result(v1.size());
    for (size_t i = 0; i < v1.size(); i++) {
        result[i] = v1[i] || v2[i];
    }
    return result;
}

struct busca_exaustiva_gpu 
{
    busca_exaustiva_gpu(const vector<FilmeProcessado>& vetor_filmes_processado, int qtd_filmes, int qtd_categorias, const vector<int>& filmes_por_categoria) {}
    __host__ __device__
    int operator()(const int &config , const vector<FilmeProcessado>&vetor_filmes_processado, const int &qtd_filmes, const int &qtd_categorias, const vector<int> &filmes_por_categoria) {
        vector<bool> horarios_disponiveis(24, false);
        vector<int> filmes_por_categoria_aux = filmes_por_categoria;
        int max_count = 0;
        for (int i = 0; i < qtd_filmes; i++){
            if (config & (1 << i)){
                if (filmes_por_categoria_aux[vetor_filmes_processado[i].categoria-1] > 0){
                    int horario_analisado = and_vectors(horarios_disponiveis, vetor_filmes_processado[i].horario);
                    // vector<bool> horario_analisado = horarios_disponiveis & vetor_filmes_processado[i].horario;
                    if ((horario_analisado != 0)) return -1;
                    filmes_por_categoria_aux[vetor_filmes_processado[i].categoria-1]--;
                    // horarios_disponiveis |= vetor_filmes_processado[i].horario;
                    horarios_disponiveis = or_vectors(horarios_disponiveis, vetor_filmes_processado[i].horario);
                    max_count += 1;
                }
                else{
                    return -1;
                }
            }
        
        }

        return max_count;
    }
};


int main(){
    int qtd_filmes, qtd_categorias;
    cin >> qtd_filmes >> qtd_categorias;

    vector<int> filmes_por_categoria(qtd_categorias, 0);
    Filme filme_vazio = {0, 0, 0};
    vector<Filme> vetor_filmes (qtd_filmes, filme_vazio);
    vector<bool> bool_vazio (24,false);
    FilmeProcessado filme_processado_vazio = {0, bool_vazio};
    vector<FilmeProcessado> vetor_filmes_processado (qtd_filmes, filme_processado_vazio);

    vector<bitset<64>> vetor_schedules;

    for (int i = 0; i < qtd_categorias; i++){
        cin >> filmes_por_categoria[i];
    }

    for (int i = 0; i < qtd_filmes; i++){
        Filme filme;
        cin >> filme.inicio >> filme.fim >> filme.categoria;
        if (filme.inicio == 0) filme.inicio = 24;
        if (filme.fim == 0) filme.fim = 24;
        if (filme.inicio < 0 || filme.fim < 0) continue;

        vetor_filmes[i] = filme;
    }

    for (int i = 0; i < qtd_filmes; i++){
        vetor_filmes_processado[i].categoria = vetor_filmes[i].categoria;
        preenche_bitset(vetor_filmes_processado[i].horario, vetor_filmes[i].inicio-1, vetor_filmes[i].fim-1);
        // for (int j = 0; j < 24; j++){
        //   cout << vetor_filmes_processado[i].horario[j] << " " ;
        // }
        // cout << endl;
    }


    thrust::device_vector<int> config_vector_cpu(pow(2, qtd_filmes), 0);
    thrust::counting_iterator<int> config_begin(0);
    thrust::counting_iterator<int> config_end(pow(2, qtd_filmes));

    thrust::device_vector<int> config_vector_gpu(config_vector_cpu);
    thrust::transform(config_begin, config_end, config_vector_gpu.begin(), busca_exaustiva_gpu(vetor_filmes_processado, qtd_filmes, qtd_categorias, filmes_por_categoria));

    thrust::host_vector<int> config_vector_cpu_final = config_vector_gpu;

    int max_count = 0;
    for (int i = 0; i < pow(2, qtd_filmes); i++){
        if (config_vector_cpu[i] > max_count){
            max_count = config_vector_cpu[i];
        }
    }

    cout << max_count << endl;
}


// g++ -Wl,-z,stack-size=4194304 exaustiva.cpp -o exaustiva
//  g++ -Wl,-z,stack-size=6000000000 -fopenmp exaustiva.cpp -o exaustiva
// user@monstrinho:~/ProjetoSupercomp$ ./exaustiva 
// nvcc -arch=sm_70 -rdc=true -o exaustiva_thrust exaustiva_thrust.cu