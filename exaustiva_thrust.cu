#include "hip/hip_runtime.h"
#include <cmath>
#include <algorithm> 
#include <iostream>
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
 // imports do thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h> 
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;
int main(){
    int N, M;
    cin >> N >> M;
    cout << N << endl;
    cout << M << endl;
    thrust::host_vector<int>filmes_por_categoria_cpu(M);
    thrust::host_vector<int> start_times_cpu(N);
    thrust::host_vector<int> end_times_cpu(N);
    thrust::host_vector<int> categories_cpu(N);

    for (int i = 0; i < M; i++){
        cin >> filmes_por_categoria_cpu[i];
    }

    for (int i = 0; i < N; i++){
        cin >> start_times_cpu[i];
        cin >> end_times_cpu[i];
        cin >> categories_cpu[i];
        if (start_times_cpu[i] == 0){
            start_times_cpu[i] = 24;
        }
        if (end_times_cpu[i] == 0){
            end_times_cpu[i] = 24;
        }
        if (start_times_cpu[i] < 0){
            continue;
        }
        if (end_times_cpu[i] < 0){
            continue;
        }
    }

    thrust::device_vector<int> start_times_gpu(start_times_cpu);
    thrust::device_vector<int> end_times_gpu(end_times_cpu);
    thrust::device_vector<int> categories_gpu(categories_cpu);
    thrust::device_vector<int> filmes_por_categoria_gpu(filmes_por_categoria_cpu);


    thrust::device_vector<int> dp(N * M, 0);

    // Inicializar a primeira linha da matriz com zeros
    thrust::fill(dp.begin(), dp.begin() + M, 0);

    // Preencher a matriz com as soluções para subproblemas menores
    // Preencher a matriz com as soluções para subproblemas menores
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
            int max_count = 0;
            for (int k = 0; k < i; k++) {
              if (categories_gpu[k] == j && end_times_gpu[k] <= start_times_gpu[i] && dp[(k*(M)) + j] + 1 <= filmes_por_categoria_gpu[j]) {
                  max_count = max(max_count, dp[(k*(M)) + j] + 1);
              } else {
                  max_count = max(max_count, dp[(k*(M)) + j]);
              }
            }
            dp[(i*(M)) + j] = max_count;
        }
    }

    // Encontrar o número máximo de filmes que podem ser assistidos
    int max_count = 0;
    int max_j = 0;
    for (int j = 0; j < M; j++) {
        if (dp[(N-1 * (M)) + j] > max_count) {
            max_count = dp[(N-1 * (M)) + j];
            max_j = j;
        }
    }



    cout << max_count << endl;
    cout << max_j << endl;
    cout << dp[(N-1 * (M)) + max_j] << endl;



    return 0;
}