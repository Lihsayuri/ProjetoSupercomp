#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <bitset>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

struct Filme{
    int inicio;
    int fim;
    int categoria;
};

struct StructSchedule{
    vector<int> filmes;
    int qtd_filmes;
};

struct FilmeProcessado{
    int categoria;
    bitset<24> horario;
};

void preenche_bitset(int &horarios_disponiveis, int inicio, int fim){
    for (int i = 0; i < 24; i++){
        if (i >= inicio && i < fim){
            horarios_disponiveis  |= (1 << i);
        }
        else if (inicio > fim && (i >= inicio || i < fim)){
            horarios_disponiveis |= (1 << i);
        }
        else if(inicio == fim){
            horarios_disponiveis |= (1 << i);
        }
    }
}

int and_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    for (size_t i = 0; i < v1.size(); i++) {
        if (v1[i] && v2[i]) {
            return 1;
        }
    }
    return 0;
}

vector<bool> or_vectors(const vector<bool>& v1, const vector<bool>& v2) {
    vector<bool> result(v1.size());
    for (size_t i = 0; i < v1.size(); i++) {
        result[i] = v1[i] || v2[i];
    }
    return result;
}

struct busca_exaustiva_gpu 
{  
    int qtd_filmes;
    int qtd_categorias;
    int* filmes_por_categoria;
    int* horario_filmes;
    int* categoria_filmes;
    busca_exaustiva_gpu(int qtd_filmes_, int qtd_categorias_, int* filmes_por_categoria_, int* horario_filmes_, int* categoria_filmes_) : 
    qtd_filmes(qtd_filmes_), qtd_categorias(qtd_categorias_), filmes_por_categoria(filmes_por_categoria_), horario_filmes(horario_filmes_), categoria_filmes(categoria_filmes_) {}
    __host__ __device__
    int operator()(const int& config) {
        int horarios_disponiveis = 0;
        int filmes_por_categoria_aux[99];
        for (int i = 0; i < qtd_categorias; i++){
            filmes_por_categoria_aux[i] = *(filmes_por_categoria+i);
        }
        int max_count = 0;
        for (int i = 0; i < qtd_filmes; i++){
            if (config & (1 << i)){
                if (filmes_por_categoria_aux[categoria_filmes[i]-1] > 0){
                    int horario_analisado = horarios_disponiveis & *(horario_filmes + i);
                    if (((horario_analisado) != 0)) return -1;
                    filmes_por_categoria_aux[categoria_filmes[i]-1]--;
                    horarios_disponiveis = horarios_disponiveis | *(horario_filmes + i);
                    max_count += 1;
                }
            }
        
        }

        return max_count;
    }
};



int main(){
    int qtd_filmes, qtd_categorias;
    cin >> qtd_filmes >> qtd_categorias;

    thrust::host_vector<int> filmes_por_categoria(qtd_categorias, 0);
    Filme filme_vazio = {0, 0, 0};
    vector<Filme> vetor_filmes (qtd_filmes, filme_vazio);

    for (int i = 0; i < qtd_categorias; i++){
        cin >> filmes_por_categoria[i];
    }

    for (int i = 0; i < qtd_filmes; i++){
        Filme filme;
        cin >> filme.inicio >> filme.fim >> filme.categoria;
        if (filme.inicio == 0) filme.inicio = 24;
        if (filme.fim == 0) filme.fim = 24;
        if (filme.inicio < 0 || filme.fim < 0) continue;

        vetor_filmes[i] = filme;
    }

    thrust::host_vector<int> categoria_filmes(qtd_filmes);
    thrust::host_vector<int> horarios_filmes_cpu(qtd_filmes); 


    for (int i = 0; i < qtd_filmes; i++){
        horarios_filmes_cpu[i] = 0;
        preenche_bitset(horarios_filmes_cpu[i], vetor_filmes[i].inicio-1, vetor_filmes[i].fim-1);
        categoria_filmes[i] = vetor_filmes[i].categoria;
    }


    thrust::device_vector<int> config_vector_gpu(pow(2, qtd_filmes));

    thrust::sequence(config_vector_gpu.begin(), config_vector_gpu.end());

    thrust::device_vector<int> categoria_filmes_gpu(categoria_filmes);
    thrust::device_vector<int> horarios_filmes_gpu(horarios_filmes_cpu);
    thrust::device_vector<int> filmes_por_categoria_gpu(filmes_por_categoria);

    thrust::transform(config_vector_gpu.begin(), config_vector_gpu.end(), config_vector_gpu.begin(), busca_exaustiva_gpu(qtd_filmes, qtd_categorias, raw_pointer_cast(filmes_por_categoria_gpu.data()), raw_pointer_cast(horarios_filmes_gpu.data()), raw_pointer_cast(categoria_filmes_gpu.data())));

    thrust::host_vector<int> config_vector_cpu_final = config_vector_gpu;

    int max_count = 0;
    for (int i = 0; i < pow(2, qtd_filmes); i++){
        if (config_vector_cpu_final[i] > max_count){
            max_count = config_vector_cpu_final[i];
        }
    }

    cout << max_count << endl;
}




// g++ -Wl,-z,stack-size=4194304 exaustiva.cpp -o exaustiva
//  g++ -Wl,-z,stack-size=6000000000 -fopenmp exaustiva.cpp -o exaustiva
// user@monstrinho:~/ProjetoSupercomp$ ./exaustiva 
// nvcc -arch=sm_70 -rdc=true -o exaustiva_thrust exaustiva_thrust.cu